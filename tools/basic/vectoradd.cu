#include "hip/hip_runtime.h"
#include "common.h"

void add_host(const int* a, const int* b, int* c, const int N) {
  for (int i = 0; i < N; ++i) {
    c[i] = a[i] + b[i];
  }
}

__global__
void add_device(const int* a, const int* b, int* c, const int N) {
  if (threadIdx.x < N) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
  }
}

int main(int argc, char *argv[]) {
  google::InitGoogleLogging(argv[0]);
  google::LogToStderr();
  const int NUM = 100;

  // Allocate memory on host
  int* a_host = new int[NUM];
  int* b_host = new int[NUM];
  int* c_host = new int[NUM];
  int* d_host = new int[NUM];

  // Fill vector a & b with random values
  for (int i = 0; i < NUM; ++i) {
    a_host[i] = rand() % 1000;
    b_host[i] = rand() % 1000;
  }
  
  // Allocate memory on device
  const int bytes = sizeof(int) * NUM;
  int* a_device = NULL;
  int* b_device = NULL;
  int* c_device = NULL;
  CUDA_CHECK(hipMalloc(&a_device, bytes));
  CUDA_CHECK(hipMalloc(&b_device, bytes));
  CUDA_CHECK(hipMalloc(&c_device, bytes));
  
  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(a_device, a_host, bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(b_device, b_host, bytes, hipMemcpyHostToDevice));

  // Call "kernel" routine to execute on GPU
  CUKERNEL_CHECK((add_device<<<1, NUM>>>(a_device, b_device, c_device, NUM)));

  // copy result from device to host
  CUDA_CHECK(hipMemcpy(d_host, c_device, bytes, hipMemcpyDeviceToHost));

  // Call host code to execute on CPU
  add_host(a_host, b_host, c_host, NUM);

  // Check the results
  for (int i = 0; i < NUM; ++i) {
    CHECK_EQ(c_host[i], d_host[i])
        << "check failed at " << i << ": " << c_host[i] << " vs " << d_host[i];
  }
  LOG(INFO) << "check passed";

  // Free memory on host
  delete[] a_host;
  delete[] b_host;
  delete[] c_host;
  delete[] d_host;  

  // free memset on device
  CUDA_CHECK(hipFree(a_device));
  CUDA_CHECK(hipFree(b_device));
  CUDA_CHECK(hipFree(c_device));

  return 0;
}
